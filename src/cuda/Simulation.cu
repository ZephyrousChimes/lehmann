#include "hip/hip_runtime.h"
#include "cuda/Simulation.cuh"

namespace Lehmann {
  __global__ void SimulateStep(
  const BankState* current,
  BankState* next,
  const int* exposure_targets,
  const float* exposure_weights,
  int K,
  int n)
  {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    BankState& me = next[idx];

    if (me.failed) return;

    float loss = 0.0f;

    // Loop through my K exposures
    for (int j = 0; j < K; ++j) {
      int target_bank_id = exposure_targets[idx * K + j];
      double weight = exposure_weights[idx * K + j];

      if (current[target_bank_id].failed) {
        loss += me.assets * weight;
      }
    }

    me.assets -= loss;

    if (me.assets < me.liabilities) {
      me.failed = true;
    }

  }
}


