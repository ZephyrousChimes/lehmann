#include "hip/hip_runtime.h"
#include "cuda/Simulation.cuh"

namespace Lehmann {
  __global__ void SimulateStep(
    BankState *banks,
    const int *exposure_targets,
    const double *exposure_weights,
    const int K,
    const int n
  ) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    BankState& me = banks[idx];

    if (me.failed) return; // already failed, skip

    float loss = 0.0f;

    // Loop through my K exposures
    for (int j = 0; j < K; ++j) {
      int target_bank_id = exposure_targets[idx * K + j];
      double weight = exposure_weights[idx * K + j];

      if (banks[target_bank_id].failed) {
        loss += me.assets * weight;  // lose proportional value
      }
    }

    me.assets -= loss;

    if (me.assets < me.liabilities) {
      me.failed = true;
    }

  }
}


