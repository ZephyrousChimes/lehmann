//
// Created by kronos on 8/3/25.
//

#include <cstdio>
#include <iostream>

#include "core/config.h"
#include "cuda/Bank.cuh"

int main() {
  Lehmann::BankState* h_banks = new Lehmann::BankState[Lehmann::NUM_BANKS];

  int *h_exposure_targets = new int[Lehmann::NUM_BANKS * Lehmann::EXPOSURE];
  float *h_exposure_weights = new float[Lehmann::NUM_BANKS * Lehmann::EXPOSURE];

  for (int i = 0; i < Lehmann::NUM_BANKS; ++i) {
    for (int j = 0; j < Lehmann::EXPOSURE; ++j) {
      int neighbor = (i + j + 1) % Lehmann::NUM_BANKS; // next few banks in sequence
      h_exposure_targets[i * Lehmann::EXPOSURE + j] = neighbor;
      h_exposure_weights[i * Lehmann::EXPOSURE + j] = 0.05f; // each exposure is 5%
    }
  }

  int* d_exposure_targets;
  float* d_exposure_weights;

  hipMalloc(&d_exposure_targets, Lehmann::NUM_BANKS * Lehmann::EXPOSURE * sizeof(int));
  hipMalloc(&d_exposure_weights, Lehmann::NUM_BANKS * Lehmann::EXPOSURE * sizeof(float));

  hipMemcpy(d_exposure_targets, h_exposure_targets, Lehmann::NUM_BANKS * Lehmann::EXPOSURE * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_exposure_weights, h_exposure_weights, Lehmann::NUM_BANKS * Lehmann::EXPOSURE * sizeof(float), hipMemcpyHostToDevice);

  for (int i = 0; i < 5; ++i) {
    std::cout << "Bank " << i << " exposed to: ";
    for (int j = 0; j < Lehmann::EXPOSURE; ++j) {
      std::cout << h_exposure_targets[i * Lehmann::EXPOSURE + j] << " ";
    }
    std::cout << "\n";
  }


  delete[] h_banks;
  delete[] d_exposure_targets;
  delete[] d_exposure_weights;
  hipFree(d_exposure_targets);
  hipFree(d_exposure_weights);

  return 0;
}